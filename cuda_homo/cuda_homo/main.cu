#include "hip/hip_runtime.h"
//
//  main.c
//  C_start
//
//  Created by Xiufeng Yang on 3/15/18.
//  Copyright © 2018 Xiufeng Yang. All rights reserved.
//

#include <stdio.h>
//#include <stdlib.h>
//#include <time.h>
#include <math.h>
//#include <fcntl.h>
//#include <time.h>
#define input_num 16673
#define input_dim 256
#define output_dim  1
#define hid_dim 256


#define rando() ((double)rand()/((double)RAND_MAX+1))




__global__ void z1_a1(int* d_x, double* d_W1, double* d_z1, double* d_a1, int m)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    double sum=0.0;
    
    for (int n = 0; n < 256; n++)
    {
        sum = sum+ d_x[m*256+n]*d_W1[n*256+index];
    }
    d_z1[m*256+index] = sum;
    d_a1[m*256+index] =1.0/(1.0+std::exp(-sum));
}

__device__ void z1_device(int* d_x, double* d_W1, double* d_z1, double* d_a1, int m)
{
    
    z1_a1 <<< 16, 16 >>> (d_x, d_W1,  d_z1,  d_a1,  m);
    hipDeviceSynchronize();
    

}


__global__ void z2_a2(double* d_z2, double* d_a2, double* d_z1, double* d_a1, double* d_W2, int m)
{
    //int index = threadIdx.x;
    double sum1=0.0;
    
    for (int n = 0; n < 256; n++)
    {
    sum1 +=d_a1[m*256+n]*d_W2[n];
    }
    d_z2[m] = sum1;
    d_a2[m] =1.0/(1.0+std::exp(-sum1));
    
    
    
}

__device__ void z2_device(double* d_z2, double* d_a2, double* d_z1, double* d_a1, double* d_W2, int m)
{
    z2_a2 <<< 1, 1 >>> (d_z2,  d_a2, d_z1, d_a1,  d_W2, m);
    hipDeviceSynchronize();
}


__global__ void W2_de(double* d_a2, double* d_Y, double* d_a1, double* d_W2, int m)
{
    // backpropagation of W2
    int index = threadIdx.x;
    double gr1=0.0;
    double lr=0.1;

 
            
    gr1=(d_a2[m]-d_Y[m])*(d_a2[m]*(1.0-d_a2[m]))*d_a1[index];
    
    d_W2[index]=d_W2[index]-lr*gr1;
            
        
    
    
}

__device__ void W2_gradient(double* d_a2, double* d_Y, double* d_a1, double* d_W2, int m)
{
    W2_de <<<1, 256>>> (d_a2, d_Y, d_a1, d_W2, m);
    hipDeviceSynchronize();
}

__global__ void W1_de(int* d_x, double* d_a2, double* d_Y, double* d_a1, double* d_W2, double* d_W1, int m )
{
    // calculate gradident of w1
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    double gr2=0.0;
    
        for (int h=0; h<256; h++)
        {
          
            gr2=(d_a2[m]-d_Y[m])*(d_a2[m]*(1.0-d_a2[m]))*d_W2[h]*(d_a1[m*256+h]*(1.0-d_a1[m*256+h]))*d_x[m*256+index];
            
            d_W1[index*256+h]=d_W1[index*256+h]-0.1*gr2;
        }
    

    
}

__device__ void W1_gradient(int* d_x, double* d_a2, double* d_Y, double* d_a1, double* d_W2, double* d_W1, int m)
{
    W1_de <<<16 , 16>>> (d_x, d_a2, d_Y, d_a1, d_W2,d_W1, m);
    hipDeviceSynchronize();
}



__global__ void training(int* d_x, double* d_Y, 
double* d_W1, double* d_z1, double* d_a1, double* d_W2, double* d_z2, double* d_a2, int* d_randput)
{ printf("shit");
    //double T_loss=0.0, Loss=0.0;
    for (int e = 0; e < 50; ++e)
    {
        double T_loss=0.0, Loss=0.0;

        //printf("%f \n", T_loss);

        for (int p = 0; p < 16673; p++)
        {
            //Loss = 0.5*(d_Y[0]-d_a2[0])*(d_Y[0]-d_a2[0]);
            //T_loss =T_loss+Loss;
            //printf("%f \n", T_loss);
            int m=d_randput[p];
            z1_device(d_x,  d_W1,  d_z1, d_a1,  m);
            z2_device( d_z2, d_a2,  d_z1,  d_a1, d_W2,m);
            W2_gradient( d_a2,  d_Y, d_a1,  d_W2, m);
            W1_gradient(d_x, d_a2, d_Y, d_a1, d_W2,d_W1, m);
            Loss = 0.5*(d_Y[m]-d_a2[m])*(d_Y[m]-d_a2[m]);
            
            T_loss =T_loss+Loss;


            
        }
     printf("loss: %f \n", (1.0/16673.0)*T_loss);

        
    }

}




int main(void)


{
    int i, j;
    //double sum1=0.0, sum2=0.0, gr1=0.0, gr2=0.0;
    //double Loss=0.0, T_loss=0.0, lr = 0.1;
    
    /* allocate memory size for training data X, Y and W1, W2, z1, z2, a1, a2 on GPU */
    //const long signed int insize = input_num*input_dim*sizeof(int);
    //const long signed int outsize = input_num*output_dim*sizeof(double);

    
      float gpu_elapsed_time_ms, cpu_elapsed_time_ms;
    int* randput=(int*)malloc(input_num*sizeof(int));
    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
// start to count execution time of GPU version
    hipEventRecord(start, 0);
    
    double* W1=(double*)malloc(input_dim*hid_dim*sizeof(double));
    
    double* W2=(double*)malloc(hid_dim*output_dim*sizeof(double));
    
    double* z1=(double*)malloc(input_num*hid_dim*sizeof(double));
    
    double* a1=(double*)malloc(input_num*hid_dim*sizeof(double));
    
    double* z2=(double*)malloc(input_num*output_dim*sizeof(double));
    
    double* a2=(double*)malloc(input_num*output_dim*sizeof(double));
 
    int* X=(int*)malloc(input_num*input_dim*sizeof(int));
    
    double* Y=(double*)malloc(input_num*output_dim*sizeof(double));
    double* h_a2=(double*)malloc(input_num*output_dim*sizeof(double));

    
    
    /* initialize training data, W1 and W2*/
    
    FILE *file;
    file=fopen("/home/yang/X.txt", "r");
    
    for(i = 0; i < input_num; i++)
    {
        for(j = 0; j < input_dim; j++)
        {
            //Use lf format specifier, %c is for character
            if (!fscanf(file, "%d", &X[i*input_dim+j]))
                break;
            // mat[i][j] -= '0';
            //printf("%d\n",X[i*input_dim+j]); //Use lf format specifier, \n is for new line
        }
        
    }
    
    FILE *file1;
    file1=fopen("/home/yang/Y.txt", "r");
    for(i = 0; i < input_num; i++)
    {
        for(j = 0; j < output_dim; j++)
        {
            //Use lf format specifier, %c is for character
            if (!fscanf(file1, "%lf", &Y[i*output_dim+j]))
                break;
            // mat[i][j] -= '0';
            //printf("%lf\n",Y[i*output_dim+j]); //Use lf format specifier, \n is for new line
        }
        
    }
    
    fclose(file);
    fclose(file1);
    
    
    for (i=0; i<input_num; i++)
    {
        randput[i]=i;
    }
    
    
    for( i= 0 ; i <input_dim; i++ )
    {    /* initialize Weight W1*/
        for( j = 0 ; j <hid_dim; j++ )
        {
            W1[i*hid_dim+j] =2.0*(rando()-0.5)*0.5;
        }
    }
    
    for( i= 0 ; i <hid_dim; i++ )
    {    /* initialize Weight W1*/
        for( j = 0 ; j <output_dim; j++ )
        {
            W2[i*output_dim+j] =2.0*(rando()-0.5)*0.5;
        }
    }
    
    // randomize the order of training data
    srand (time(NULL));
    for(int k=input_num-1; k>=0; k--)
    {
        j=rand() % (k+1);
        int temp=randput[k];
        randput[k]=randput[j];
        randput[j]=temp;
        
    }
    //printf("%f \n",(0.5/16673)*T_loss);
    //Loss=0.0;
    //T_loss=0.0;
    
    /* Allocate matrices d_x, d_y, d_W1, d_W2, d_z1, d_a1, d_z2, d_a2 on device*/
    
    int *d_x;
    double *d_Y;
    double *d_W1, *d_W2, *d_z1, *d_a1, *d_z2, *d_a2;
    int *d_randput;
    //const long signed int insize = input_num*input_dim*sizeof(int);
    //const long signed int outsize = input_num*output_dim*sizeof(double);

    hipMalloc(&d_x, input_num*input_dim*sizeof(int));
    hipMalloc(&d_Y,input_num*output_dim*sizeof(double));
    hipMalloc(&d_W1, input_dim*hid_dim*sizeof(double));
    hipMalloc(&d_z1, input_num*hid_dim*sizeof(double));
    hipMalloc(&d_a1, input_num*hid_dim*sizeof(double));
    hipMalloc(&d_W2, hid_dim*output_dim*sizeof(double));
    hipMalloc(&d_z2, input_num*output_dim*sizeof(double));
    hipMalloc(&d_a2, input_num*output_dim*sizeof(double));
    hipMalloc(&d_randput, input_num*sizeof(int));
    
    
    /* copy memory allocated for matrices from host to device*/
    hipMemcpy(d_x, X, input_num*input_dim*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, input_num*output_dim*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_W1, W1, input_dim*hid_dim*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_z1, z1, input_num*input_dim*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_a1, a1, input_num*input_dim*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_W2, W2, hid_dim*output_dim*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_z2, z2, input_num*output_dim*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_a2, a2, input_num*output_dim*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_randput, randput, input_num*sizeof(int),hipMemcpyHostToDevice);



    
    /*start the kernel to perform stochastic gradient descent*/
    
    
    training <<< 1, 1 >>> (d_x, d_Y, d_W1, d_z1, d_a1, d_W2, d_z2, d_a2, d_randput);

    hipMemcpy(h_a2, d_a2,input_num*output_dim*sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed training  on GPU: %f ms.\n\n",gpu_elapsed_time_ms);
    for (int i = 0; i < 100; i++)
{
 
printf("Predicted : %f True : %f  \n",  h_a2[i],  Y[i]);
	}

    
    hipFree(d_x);
    hipFree(d_Y);
    hipFree(d_W1);
    hipFree(d_W2);
    hipFree(d_z1);
    hipFree(d_z2);
    hipFree(d_a1);
    hipFree(d_a2);
    hipFree(d_randput);
     
    free(W1);
    free(W2);
    free(z1);
    free(a1);
    free(a2);
    //free(X);
    //free(Y);
    free(randput);
    
    
    
    return 1 ;
    
}
